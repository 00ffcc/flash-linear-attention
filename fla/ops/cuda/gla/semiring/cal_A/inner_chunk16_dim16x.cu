#include <stdio.h>
#include <hip/hip_runtime.h>

#define CEIL_DIV(M, N) (((M) + (N)-1) / (N))

#define FLOAT4(pointer) (reinterpret_cast<float4*>(&(pointer))[0])

__global__ void fwd_inner_chunk16_dim16x(int batchSize, int M, int N_K, 
                                     float *Q, float *K, float *G_K, 
                                     float *QK
                                    ) {

  // Batch index
  const uint batchIdx = blockIdx.x;
  // allocate buffer for current block in fast shared mem
  __shared__ float Q_tile[16][16];
  __shared__ float K_tile[16][16];
  __shared__ float G_tile[16][16];
  __shared__ float G_tile_trans[16][16];

  const uint threadCol = threadIdx.x % 16;
  const uint threadRow = threadIdx.x / 16;

  int K_Stride = M * N_K;

  // Adjust the pointers for batch and matrix size
  Q += batchIdx * K_Stride;
  K += batchIdx * K_Stride;
  G_K += batchIdx * K_Stride;
  QK += batchIdx * M * M;
  
  float tmp = 0.0;

  for (int bkIdx = 0; bkIdx < N_K; bkIdx += 16) {
    Q_tile[threadRow][threadCol] = Q[threadRow * N_K + threadCol];
    K_tile[threadRow][threadCol] = K[threadRow * N_K + threadCol];
    float tmp_gk = G_K[threadRow * N_K + threadCol];
    G_tile[threadRow][threadCol] = tmp_gk;
    G_tile_trans[threadCol][threadRow] = tmp_gk;

    __syncthreads();

    Q += 16;
    K += 16;
    G_K += 16;
    
    if(threadCol <= threadRow){
        for (int dotIdx = 0; dotIdx < 16; ++dotIdx) {
            // avoid bank conflict?
            float exp_term = expf(G_tile[threadRow][dotIdx] - G_tile_trans[dotIdx][threadCol]);
            tmp += Q_tile[threadRow][dotIdx] * K_tile[threadCol][dotIdx] * exp_term;
        }
    }
    __syncthreads();    
  }  

  if(threadCol <= threadRow){
    QK[threadRow * M + threadCol] = tmp;
  }  
  else{
    QK[threadRow * M + threadCol] = 0.0;
  }
    
}

void run_fwd_inner_chunk16_dim16x(int batchSize, int M, int N_K, 
                                float *Q, float *K, float *gK, float *QK
                            ) {  
  dim3 gridDim(batchSize); 
  dim3 blockDim(256);
  fwd_inner_chunk16_dim16x<<<gridDim, blockDim>>>(batchSize, M, N_K, Q, K, gK, QK); 
}


__global__ void bwd_inner_chunk16_dim16x(int batchSize, int M, int N_K, 
                                     float *Q, float *K, float *G, 
                                     float *DQK, float *DQ, float *DK, 
                                     float *DG
                                    ) {

  // Batch index
  const uint batchIdx = blockIdx.x;
  
  // allocate buffer for current block in fast shared mem
  __shared__ float Q_tile[16][16];
  __shared__ float QK_tile[16][16];
  __shared__ float K_tile[16][16];
  __shared__ float G_tile[16][16];
  __shared__ float G_tile_trans[16][16];
  
  const uint threadCol = threadIdx.x % 16;
  const uint threadRow = threadIdx.x / 16;

  int K_Stride = M * N_K;

  Q += batchIdx * K_Stride;
  DQ += batchIdx * K_Stride;
  K += batchIdx * K_Stride;
  DK += batchIdx * K_Stride;
  G += batchIdx * K_Stride;
  DG += batchIdx * K_Stride;
  
  DQK += batchIdx * M * M;
  QK_tile[threadRow][threadCol] = (threadCol <= threadRow) ? DQK[threadRow * M + threadCol] : 0.0;
  __syncthreads();

  for (int bkIdx = 0; bkIdx < N_K; bkIdx += 16) {
    Q_tile[threadRow][threadCol] = Q[threadRow * N_K + threadCol];
    K_tile[threadRow][threadCol] = K[threadRow * N_K + threadCol];
    float tmp_gk = G[threadRow * N_K + threadCol];
    G_tile[threadRow][threadCol] = tmp_gk;
    // G_tile_trans[threadCol][threadRow] = tmp_gk;

    __syncthreads();

    float threadResults_dK = 0;
    float threadResults_dQ = 0;
    
    for(uint dotIdx = threadRow; dotIdx < 16; dotIdx += 1){
          float tmp =  QK_tile[dotIdx][threadRow] * expf(G_tile[dotIdx][threadCol] - G_tile[threadRow][threadCol]) * Q_tile[dotIdx][threadCol];
          threadResults_dK += tmp;                  
    }
    
    for(uint dotIdx = 0; dotIdx <= threadRow;  dotIdx += 1){
      float tmp = QK_tile[threadRow][dotIdx] * expf(G_tile[threadRow][threadCol] - G_tile[dotIdx][threadCol]) * K_tile[dotIdx][threadCol];                                         
      threadResults_dQ += dotIdx <= threadRow? tmp: 0;                       
    }

    __syncthreads();    
    DQ[threadRow * N_K + threadCol] = threadResults_dQ;
    DK[threadRow * N_K + threadCol] = threadResults_dK;
    DG[threadRow * N_K + threadCol] = threadResults_dQ * Q_tile[threadRow][threadCol] - threadResults_dK * K_tile[threadRow][threadCol];
    Q += 16;
    K += 16;
    G += 16;
    DQ += 16;
    DK += 16;
    DG += 16;
    __syncthreads();
  }  
}

void run_bwd_inner_chunk16_dim16x(int batchSize, int M, int N_K, 
                                float *Q, float *K, float *G, float *DQK,
                                float *DQ, float *DK, float *DG
                            ) {  
  dim3 gridDim(batchSize); 
  dim3 blockDim(256);
  bwd_inner_chunk16_dim16x<<<gridDim, blockDim>>>(batchSize, M, N_K, Q, K, G, DQK, DQ, DK, DG); 
}

